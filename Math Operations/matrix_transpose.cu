#include "hip/hip_runtime.h"
%%writefile matrix_transpose_row_wise.cu

#include<stdio.h>
#include<stdlib.h>

void print_2d_matrix(int *a, int r, int c) {
  for (int i = 0; i < r; i++) {
    for (int j = 0; j < c; j++) {
      printf("%d ", a[i*c+j]);
    }
    printf("\n");
  }
}

__global__ void transpose_row_wise_per_thread(int *a, int *out, int r, int c) {
  // Extract threadid
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i=0; i < c; i++) {
    out[i*c+idx] = a[idx*c+i];
  }
}

int main() {
  int R = 100;
  int C = 100;
  int *a, *d_a, *d_out, *out;

  a = (int *)malloc(sizeof(int) * R * C);
  out = (int *)malloc(sizeof(int) * R * C);

  for (int i = 0; i < R; i++) {
    for (int j = 0; j < C; j++) {
      a[(i*C) + j] = (100^i)*(10^j);
    }
  }

  // Move the memory to GPU

  hipMalloc((void **)&d_a, R * C * sizeof(int));
  hipMalloc((void **)&d_out, R * C * sizeof(int));

  hipMemcpy(d_a, a, R * C * sizeof(int), hipMemcpyHostToDevice);

  printf("Assign 2d memory col memory for the GPU pointers.\n");

  transpose_row_wise_per_thread<<<1,R>>>(d_a, d_out, R, C);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
  }

  hipMemcpy(out, d_out, R * C * sizeof(int), hipMemcpyDeviceToHost);

  printf("Out from GPU .. \n");


  for (int i=0; i<R; i++){
    for (int j =0; j <C; j++) {
      if (a[i*C+j] != out[j*C+i]) {
        printf("Incorrect transpose !!!\n");
        print_2d_matrix(out, R, C);
        print_2d_matrix(a, R, C);
      }
    }
  }

  hipFree(d_a);
  hipFree(d_out);

  free(out);
  free(a);

  return 0;
}