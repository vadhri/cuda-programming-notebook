#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 200,000 ns.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void init(int * a, int * b, int * c) {
    int tid = blockIdx.x * blockDim.x * threadIdx.x;

    if ( tid < N ) {
        c[tid] = 0;
        a[tid] = 2;
        b[tid] = 1;
    }
}


__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x * threadIdx.x;

    if ( tid < N )
        c[tid] = 2 * a[tid] + b[tid];
}

int main()
{
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector
    int deviceId;
    int numberOfSMs;
    hipError_t addVectorsErr;
    hipError_t asyncErr;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

    int threadsPerBlock = 128;
    int numberOfBlocks = 32*numberOfSMs;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    init <<< numberOfBlocks, threadsPerBlock >>> ( a, b, c );
    saxpy <<< numberOfBlocks, threadsPerBlock >>> ( a, b, c );

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  hipMemPrefetchAsync(c, size, hipCpuDeviceId);

  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
